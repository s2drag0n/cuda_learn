
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_from_gpu()
{
    printf("Hello world from the GPU\n");
}

int main(void)
{
  hello_from_gpu<<<2, 2>>>();
  hipDeviceSynchronize();

  return 0;
}